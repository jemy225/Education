#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>



__global__ void addKernel(int *a,  int *b,  int *c)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
	printf("%d,%d,%d\r\n", a[i],b[i], c[i]);
}

int main()
{
	int arraySize = 5;
	int* a = 0;
	int* b = 0;
	int* c = 0;
	hipMallocManaged(&a, arraySize * sizeof(int));
	hipMallocManaged(&b, arraySize * sizeof(int));
	hipMallocManaged(&c, arraySize * sizeof(int));
	for (int n = 0;n < arraySize;n++)
	{
		a[n] = n + 1;
		b[n] = 10 * (n + 1);
	}
	addKernel << <1, arraySize >> > (a, b, c);
	hipDeviceSynchronize();

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	hipDeviceReset();
	getchar();
	return 0;
}
